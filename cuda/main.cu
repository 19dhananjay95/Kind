#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>
#include <thrust/scan.h>

typedef  int64_t i64;
typedef uint64_t u64;
typedef double   f64;

// ::::::::::
// :: Node ::
// ::::::::::

__host__ __device__
u64 new_node(u64 kind, i64 a_dist, u64 a_slot, i64 b_dist, u64 b_slot, i64 c_dist, u64 c_slot) {
  return (kind << 54)
      | (a_slot << 52) | ((u64)(a_dist + 32768) << 36)
      | (b_slot << 34) | ((u64)(b_dist + 32768) << 18)
      | (c_slot << 16) | ((u64)(c_dist + 32768) <<  0);
}

__host__ __device__
u64 get_kind(u64 node) {
  return (node >> 54) & 0xFF;
}

__host__ __device__
i64 get_dist(u64 node, u64 slot) {
  return (i64)((node >> (36 - slot * 18)) & 0xFFFF) - 32768;
}

__host__ __device__
u64 get_slot(u64 node, u64 slot) {
  return ((node >> (52 - slot * 18)) & 0x3);
}

__host__ __device__
u64 inc_port(u64 node, u64 slot, i64 delta) {
  return (u64)((i64)node + (delta << (36 - slot * 18)));
}

__host__ __device__
u64 inc_ports(u64 node, i64 delta) {
  return (u64)((i64)node + (delta << 36) + (delta << 18) + delta);
}

__host__ __device__
u64 set_port(u64 node, u64 slot, i64 new_dist, u64 new_slot) {
  return (node & ~((u64)0x3FFFF << (36 - slot * 18))) | (((new_slot << 16) | (u64)(new_dist + 32768)) << (36 - slot * 18));
}

__host__ __device__
u64 eql(u64 a, u64 b) {
  return a == b;
}

__host__ __device__
f64 get_force(u64 node) {
  i64 x = get_dist(node, 0);
  i64 y = get_dist(node, 1);
  i64 z = get_dist(node, 2);
  return (f64)((x < 0 ? -1 : 1) * x * x + (y < 0 ? -1 : 1) * y * y + (z < 0 ? -1 : 1) * z * z);
}

__constant__
const u64 air = 0x8000600028000; // new_node(0, 0,0, 0,1, 0,2)

// :::::::::
// :: Net ::
// :::::::::

struct Alloc {
  u64 indxs[4];
};

__host__ __device__
bool alloc4(u64 *net, u64 len, u64 i, u64 *indxs) {
  u64 k = 0, n, a;
  u64 j = 0;
  do {
    k = k + 1;
    n = i + ((k % 2) * 2 - 1) * (k / 2);
    a = n < len ? net[n] : 0;
    if (eql(a, air)) {
      indxs[j++] = n;
    }
  } while (k < 32 && j < 4);
  return j == 4;
}

__host__ __device__
void link(u64* net, u64 len, u64 a_indx, u64 a_slot, u64 b_indx, u64 b_slot) {
  net[a_indx] = set_port(net[a_indx], a_slot, b_indx - a_indx, b_slot);
  net[b_indx] = set_port(net[b_indx], b_slot, a_indx - b_indx, a_slot);
}

__host__ __device__
void unlink(u64 *net, u64 len, u64 a_indx, u64 a_slot) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, a_slot) + a_indx;
  u64 b_slot = get_slot(a_node, a_slot);
  u64 b_node = net[b_indx];
  if (get_dist(b_node, b_slot) + b_indx == a_indx && get_slot(b_node, b_slot) == a_slot) {
    net[a_indx] = set_port(a_node, a_slot, 0, a_slot);
    net[b_indx] = set_port(b_node, b_slot, 0, b_slot);
  }
}

__host__ __device__
u64 redex_type(u64* net, u64 len, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, 0) + a_indx;
  u64 b_node = net[b_indx];
  if (get_slot(a_node, 0) == 0 && (get_dist(a_node, 0) + get_dist(b_node, 0)) == 0 && !eql(a_node, air)) {
    return get_kind(a_node) == get_kind(b_node) ? 1 : 2;
  } else {
    return 0;
  }
};

__host__ __device__
bool rewrite(u64* net, u64 len, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = a_indx + get_dist(a_node, 0);
  u64 b_node = net[b_indx];
  if (redex_type(net, len, a_indx) == 0) return false;
  if (get_kind(a_node) == get_kind(b_node)) {
    u64 a1_indx = get_dist(net[a_indx], 1) + a_indx;
    u64 a1_slot = get_slot(net[a_indx], 1);
    u64 b1_indx = get_dist(net[b_indx], 1) + b_indx;
    u64 b1_slot = get_slot(net[b_indx], 1);
    link(net, len, a1_indx, a1_slot, b1_indx, b1_slot);
    u64 a2_indx = get_dist(net[a_indx], 2) + a_indx;
    u64 a2_slot = get_slot(net[a_indx], 2);
    u64 b2_indx = get_dist(net[b_indx], 2) + b_indx;
    u64 b2_slot = get_slot(net[b_indx], 2);
    link(net, len, a2_indx, a2_slot, b2_indx, b2_slot);
  } else {
    u64 indxs[4] = {0, 0, 0, 0};
    if (!alloc4(net, len, (a_indx + b_indx) / 2, indxs)) return false;
    u64 c_indx = indxs[0];
    u64 d_indx = indxs[1];
    u64 e_indx = indxs[2];
    u64 f_indx = indxs[3];
    net[c_indx] = new_node(get_kind(b_node), 0,0, f_indx - c_indx, 1, e_indx - c_indx, 1); 
    net[d_indx] = new_node(get_kind(b_node), 0,0, f_indx - d_indx, 2, e_indx - d_indx, 2); 
    net[e_indx] = new_node(get_kind(a_node), 0,0, c_indx - e_indx, 2, d_indx - e_indx, 2);
    net[f_indx] = new_node(get_kind(a_node), 0,0, c_indx - f_indx, 1, d_indx - f_indx, 1);
    link(net, len, c_indx, 0, get_dist(net[a_indx],1) + a_indx, get_slot(net[a_indx],1));
    link(net, len, d_indx, 0, get_dist(net[a_indx],2) + a_indx, get_slot(net[a_indx],2));
    link(net, len, e_indx, 0, get_dist(net[b_indx],2) + b_indx, get_slot(net[b_indx],2));
    link(net, len, f_indx, 0, get_dist(net[b_indx],1) + b_indx, get_slot(net[b_indx],1));
  }
  for (int slot = 0; slot < 3; slot++) {
    unlink(net, len, a_indx, slot);
    unlink(net, len, b_indx, slot);
  }
  net[a_indx] = air;
  net[b_indx] = air;
  return true;
}

__host__ __device__
void move(u64 *net, u64 len, u64 a_indx, u64 b_indx) {
  u64 a_node = net[a_indx];
  u64 b_node = net[b_indx];
  net[b_indx] = inc_ports(a_node, -(b_indx - a_indx));
  net[a_indx] = inc_ports(b_node, -(a_indx - b_indx));
  for (u64 slot = 0; slot < 3; ++slot) {
    u64 a_dist = get_dist(a_node, slot);
    u64 a_slot = get_slot(a_node, slot);
    u64 b_dist = get_dist(b_node, slot);
    u64 b_slot = get_slot(b_node, slot);
    u64 c_indx = a_dist == 0 ? b_indx : a_dist == b_indx - a_indx ? a_indx : a_indx + a_dist;
    u64 d_indx = b_dist == 0 ? a_indx : b_dist == a_indx - b_indx ? b_indx : b_indx + b_dist;
    net[c_indx] = inc_port(net[c_indx], a_slot, b_indx - a_indx);
    net[d_indx] = inc_port(net[d_indx], b_slot, a_indx - b_indx);
  }
}

__host__ __device__
void chill(u64 *net, u64 len) {
  for (u64 i = 0; i < len - 1; i += 2) {
    if (get_force(net[i]) > get_force(net[i + 1])) {
      move(net, len, i, i + 1);
    }
  }
  for (u64 i = 1; i < len - 1; i += 2) {
    if (get_force(net[i]) > get_force(net[i + 1])) {
      move(net, len, i, i + 1);
    }
  }
}

__host__ __device__
bool is_valid(u64 *net, u64 len) {
  for (u64 a_indx = 0; a_indx < len; ++a_indx) {
    for (u64 a_slot = 0; a_slot < 3; ++a_slot) {
      u64 a_node = net[a_indx];
      u64 b_indx = get_dist(a_node, a_slot) + a_indx;
      u64 b_slot = get_slot(a_node, a_slot);
      u64 b_node = net[b_indx];
      if (get_dist(b_node,b_slot) != a_indx - b_indx || get_slot(b_node,b_slot) != a_slot) {
        return false;
      }
    }
  }
  return true;
}

std::vector<u64> redexes(u64 *net, u64 len) {
  std::vector<u64> redexes;
  for (u64 a_indx = 0; a_indx < len; ++a_indx) {
    u64 b_indx = get_dist(net[a_indx], 0) + a_indx;
    if (a_indx <= b_indx && redex_type(net, len, a_indx) > 0) {
      redexes.push_back(a_indx);
    }
  }
  return redexes;
}

u64 reduce_pass(u64 *net, u64 len) {
  std::vector<u64> rdx = redexes(net, len);
  u64 rwt = 0;
  for (u64 i = 0; i < rdx.size(); ++i) {
    if (rewrite(net, len, rdx[i]))  {
      rwt += 1;
    }
  }
  return rwt;
}

// ::::::::::
// :: Misc ::
// ::::::::::

std::string show_slot(u64 node, u64 slot) {
  std::string str;
  str.append(std::to_string(get_dist(node, slot)));
  switch (get_slot(node, slot)) {
    case 0: str.append("a"); break;
    case 1: str.append("b"); break;
    case 2: str.append("c"); break;
  }
  return str;
}

std::string show_node(u64 node) {
  std::string str;
  str.append(std::to_string(get_kind(node)));
  for (int slot = 0; slot < 3; ++slot) {
    str.append(slot > 0 ? " " : "[");
    str.append(show_slot(node, slot));
  }
  str.append("] {");
  str.append(std::to_string(get_force(node)));
  str.append("}");
  return str;
}

std::string plot_nums(std::vector<f64> &nums, std::vector<u64> &cols) {
  std::string str;
  for (uint i = 0; i < nums.size(); ++i) {
    str.append(cols[i] == 0 ? "\x1b[33m" : cols[i] == 1 ? "\x1b[32m" : "\x1b[31m");
    switch ((u64)(floor(fmax(fmin(nums[i],(f64)1),(f64)0) * 8))) {
      case 0: str.append(","); break;
      case 1: str.append("▁"); break;
      case 2: str.append("▂"); break;
      case 3: str.append("▃"); break;
      case 4: str.append("▄"); break;
      case 5: str.append("▅"); break;
      case 6: str.append("▆"); break;
      case 7: str.append("▇"); break;
      case 8: str.append("█"); break;
    }
    str.append("\x1b[0m");
    if (i % 128 == 127 && i < nums.size() - 1) {
      str.append("\n");
    }
  }
  return str;
};

void print_net(u64 *net, u64 len, bool show_nodes, bool show_stats, bool show_heatmap) {
  for (u64 i = 0; i < len; ++i) {
    if (show_nodes && !eql(net[i], air)) {
      std::cout << i << " - " << show_node(net[i]) << std::endl;
    }
  }
  if (show_stats) {
    std::cout << "Valid: " << is_valid(net, len) << std::endl;
  }
  if (show_heatmap) {
    std::vector<f64> nums;
    std::vector<u64> cols;
    for (u64 i = 0; i < len; ++i) {
      nums.push_back(eql(net[i], air) ? 0 : 1.0 / 8.0 + sqrt(abs(get_force(net[i]))) / 64.0);
      cols.push_back(redex_type(net, len, i));
    }
    std::cout << plot_nums(nums, cols) << std::endl;
  }
}

void print_nums(u64 *vec, u64 len) {
  for (u64 i = 0; i < len; ++i) {
    std::cout << vec[i] << " ";
  }
  std::cout << std::endl;
}

__global__
void expand(u64 *src, u64 *dst) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  u64 node = src[i];
  dst[i * 2 + 0] = new_node(get_kind(node),
    get_dist(node, 0) * 2, get_slot(node, 0),
    get_dist(node, 1) * 2, get_slot(node, 1),
    get_dist(node, 2) * 2, get_slot(node, 2));
  dst[i * 2 + 1] = air;
}

__global__
void shrink(u64 *src, u64 *dst, u64 *mov) {
  u64 src_indx = blockIdx.x * blockDim.x + threadIdx.x;
  u64 dst_indx = mov[src_indx];
  u64 node = src[src_indx]; 
  if (!eql(node, air)) {
    u64 x_dst_indx = mov[get_dist(node, 0) + src_indx];
    u64 y_dst_indx = mov[get_dist(node, 1) + src_indx];
    u64 z_dst_indx = mov[get_dist(node, 2) + src_indx];
    dst[dst_indx] = new_node(get_kind(node),
      (i64)x_dst_indx - (i64)dst_indx, get_slot(node, 0),
      (i64)y_dst_indx - (i64)dst_indx, get_slot(node, 1),
      (i64)z_dst_indx - (i64)dst_indx, get_slot(node, 2));
  }
}

// ::::::::::
// :: Main ::
// ::::::::::

struct is_node : public thrust::unary_function<u64,u64> {
  __host__ __device__ u64 operator()(u64 node) { return eql(node, air) ? 0 : 1; }
};

const std::vector<u64> ex = {0x0028000a00f08000,0x0028001200b8803b,0x0028001a006c7fff,0x0008001a00207fff,0x0007fff200108001,0x0027fff200088001,0x0027fff600048003,0x0017ffe5fffd8001,0x0017ffc9fffc8002,0x0027ffd600068001,0x0027ffe5fffe7fff,0x0008001a00217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0008001a00217fe5,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0017fd26000e8001,0x00280012002a7fff,0x00280012001c7fff,0x00080015fff47fff,0x0007fff200108001,0x0027fff6000c8001,0x0027fff600068001,0x00280015fffe7fff,0x0017ffc5fff47fff,0x0017ff9600008001,0x0027fffa00018000,0x0017ff6a00048001,0x0027fff600017fff,0x0027fc5200057fc4,0x0017fffa00048001,0x0027fff600017fff};  

int main(void) {

  // Creates net on host
  thrust::host_vector<u64> h_net(256);
  thrust::host_vector<u64> h_indx(256);
  thrust::fill(h_net.begin(), h_net.begin() + h_net.size(), air);
  thrust::fill(h_indx.begin(), h_indx.begin() + h_indx.size(), 0);
  for (int i = 0; i < ex.size(); ++i) h_net[i] = ex[i];

  // Sends to GPU
  thrust::device_vector<u64> d_net0 = h_net;
  thrust::device_vector<u64> d_net1(d_net0.size());
  thrust::device_vector<u64> d_indx(d_net0.size());
  
  // Expands
  thrust::fill(d_net1.begin(), d_net1.end(), air);
  expand<<<8,16>>>(thrust::raw_pointer_cast(&d_net0[0]), thrust::raw_pointer_cast(&d_net1[0]));

  // Sends to CPU & prints
  h_net = d_net1;
  print_net(&h_net[0], h_net.size(), true, true, true);

  // Shrinks
  thrust::fill(d_indx.begin(), d_indx.end(), 0);
  thrust::transform_exclusive_scan(d_net1.begin(), d_net1.end(), d_indx.begin(), is_node(), 0, thrust::plus<u64>());
  thrust::fill(d_net0.begin(), d_net0.end(), air);
  shrink<<<8,16>>>(thrust::raw_pointer_cast(&d_net1[0]), thrust::raw_pointer_cast(&d_net0[0]), thrust::raw_pointer_cast(&d_indx[0]));
  
  // Sends to CPU & prints
  h_net = d_net0;
  print_net(&h_net[0], h_net.size(), true, true, true);

  return 0;
}
