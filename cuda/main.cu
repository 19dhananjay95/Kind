#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>
#include <thrust/scan.h>

typedef  int64_t i64;
typedef uint64_t u64;
typedef double   f64;

// ::::::::::
// :: Node ::
// ::::::::::

// type=0 is a normal node
// type=1 is a duplicating node
// type=2 is a wire
__host__ __device__
u64 new_node(u64 kind, i64 a_dist, u64 a_slot, i64 b_dist, u64 b_slot, i64 c_dist, u64 c_slot) {
  return (kind << 54)
      | (a_slot << 52) | ((u64)(a_dist + 32768) << 36)
      | (b_slot << 34) | ((u64)(b_dist + 32768) << 18)
      | (c_slot << 16) | ((u64)(c_dist + 32768) <<  0);
}

__constant__
const u64 air = 0x8000600028000; // new_node(0, 0,0, 0,1, 0,2)

__host__ __device__ 
u64 set_type(u64 node, u64 type) {
  return (node & ~((u64)0x3 << 62)) | (type << 62);
}

__host__ __device__
u64 get_type(u64 node) {
  return (node >> 62) & 0x3;
}

__host__ __device__
u64 get_kind(u64 node) {
  return (node >> 54) & 0xFF;
}

__host__ __device__
i64 get_dist(u64 node, u64 slot) {
  return (i64)((node >> (36 - slot * 18)) & 0xFFFF) - 32768;
}

__host__ __device__
u64 get_slot(u64 node, u64 slot) {
  return ((node >> (52 - slot * 18)) & 0x3);
}

__host__ __device__
u64 inc_port(u64 node, u64 slot, i64 delta) {
  return (u64)((i64)node + (delta << (36 - slot * 18)));
}

__host__ __device__
u64 inc_ports(u64 node, i64 delta) {
  return (u64)((i64)node + (delta << 36) + (delta << 18) + delta);
}

__host__ __device__
u64 set_port(u64 node, u64 slot, i64 new_dist, u64 new_slot) {
  return (node & ~((u64)0x3FFFF << (36 - slot * 18))) | (((new_slot << 16) | (u64)(new_dist + 32768)) << (36 - slot * 18));
}

__host__ __device__
u64 eql(u64 a, u64 b) {
  return a == b;
}

__host__ __device__
f64 get_force(u64 node) {
  i64 x = get_dist(node, 0);
  i64 y = get_dist(node, 1);
  i64 z = get_dist(node, 2);
  return (f64)((x < 0 ? -1 : 1) * x * x + (y < 0 ? -1 : 1) * y * y + (z < 0 ? -1 : 1) * z * z);
}

__host__ __device__
u64 get_redex_type(u64 a_node, u64 b_node) {
  if (!eql(a_node, air) && !eql(b_node, air) && get_dist(a_node, 0) + get_dist(b_node, 0) == 0 && get_slot(a_node, 0) == 0) {
    return get_kind(a_node) == get_kind(b_node) ? 1 : 2;
  } else {
    return 0;
  }
}

// :::::::::
// :: Net ::
// :::::::::

struct Alloc {
  u64 indxs[4];
};

__host__ __device__
bool alloc4(u64 *net, u64 len, u64 i, u64 *indxs) {
  u64 k = 0, n, a;
  u64 j = 0;
  do {
    k = k + 1;
    n = i + ((k % 2) * 2 - 1) * (k / 2);
    a = n < len ? net[n] : 0;
    if (eql(a, air)) {
      indxs[j++] = n;
    }
  } while (k < 32 && j < 4);
  return j == 4;
}

__host__ __device__
void link(u64* net, u64 len, u64 a_indx, u64 a_slot, u64 b_indx, u64 b_slot) {
  net[a_indx] = set_port(net[a_indx], a_slot, b_indx - a_indx, b_slot);
  net[b_indx] = set_port(net[b_indx], b_slot, a_indx - b_indx, a_slot);
}

__host__ __device__
void unlink(u64 *net, u64 len, u64 a_indx, u64 a_slot) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, a_slot) + a_indx;
  u64 b_slot = get_slot(a_node, a_slot);
  u64 b_node = net[b_indx];
  if (get_dist(b_node, b_slot) + b_indx == a_indx && get_slot(b_node, b_slot) == a_slot) {
    net[a_indx] = set_port(a_node, a_slot, 0, a_slot);
    net[b_indx] = set_port(b_node, b_slot, 0, b_slot);
  }
}

__host__ __device__
u64 redex_type(u64* net, u64 len, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, 0) + a_indx;
  u64 b_node = net[b_indx];
  if (get_slot(a_node, 0) == 0 && (get_dist(a_node, 0) + get_dist(b_node, 0)) == 0 && !eql(a_node, air)) {
    return get_kind(a_node) == get_kind(b_node) ? 1 : 2;
  } else {
    return 0;
  }
};

__host__ __device__
bool rewrite(u64* net, u64 len, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = a_indx + get_dist(a_node, 0);
  u64 b_node = net[b_indx];
  if (redex_type(net, len, a_indx) == 0) return false;
  if (get_kind(a_node) == get_kind(b_node)) {
    u64 a1_indx = get_dist(net[a_indx], 1) + a_indx;
    u64 a1_slot = get_slot(net[a_indx], 1);
    u64 b1_indx = get_dist(net[b_indx], 1) + b_indx;
    u64 b1_slot = get_slot(net[b_indx], 1);
    link(net, len, a1_indx, a1_slot, b1_indx, b1_slot);
    u64 a2_indx = get_dist(net[a_indx], 2) + a_indx;
    u64 a2_slot = get_slot(net[a_indx], 2);
    u64 b2_indx = get_dist(net[b_indx], 2) + b_indx;
    u64 b2_slot = get_slot(net[b_indx], 2);
    link(net, len, a2_indx, a2_slot, b2_indx, b2_slot);
  } else {
    u64 indxs[4] = {0, 0, 0, 0};
    if (!alloc4(net, len, (a_indx + b_indx) / 2, indxs)) return false;
    u64 c_indx = indxs[0];
    u64 d_indx = indxs[1];
    u64 e_indx = indxs[2];
    u64 f_indx = indxs[3];
    net[c_indx] = new_node(get_kind(b_node), 0,0, f_indx - c_indx, 1, e_indx - c_indx, 1); 
    net[d_indx] = new_node(get_kind(b_node), 0,0, f_indx - d_indx, 2, e_indx - d_indx, 2); 
    net[e_indx] = new_node(get_kind(a_node), 0,0, c_indx - e_indx, 2, d_indx - e_indx, 2);
    net[f_indx] = new_node(get_kind(a_node), 0,0, c_indx - f_indx, 1, d_indx - f_indx, 1);
    link(net, len, c_indx, 0, get_dist(net[a_indx],1) + a_indx, get_slot(net[a_indx],1));
    link(net, len, d_indx, 0, get_dist(net[a_indx],2) + a_indx, get_slot(net[a_indx],2));
    link(net, len, e_indx, 0, get_dist(net[b_indx],2) + b_indx, get_slot(net[b_indx],2));
    link(net, len, f_indx, 0, get_dist(net[b_indx],1) + b_indx, get_slot(net[b_indx],1));
  }
  for (int slot = 0; slot < 3; slot++) {
    unlink(net, len, a_indx, slot);
    unlink(net, len, b_indx, slot);
  }
  net[a_indx] = air;
  net[b_indx] = air;
  return true;
}

__host__ __device__
void move(u64 *net, u64 len, u64 a_indx, u64 b_indx) {
  u64 a_node = net[a_indx];
  u64 b_node = net[b_indx];
  net[b_indx] = inc_ports(a_node, -(b_indx - a_indx));
  net[a_indx] = inc_ports(b_node, -(a_indx - b_indx));
  for (u64 slot = 0; slot < 3; ++slot) {
    u64 a_dist = get_dist(a_node, slot);
    u64 a_slot = get_slot(a_node, slot);
    u64 b_dist = get_dist(b_node, slot);
    u64 b_slot = get_slot(b_node, slot);
    u64 c_indx = a_dist == 0 ? b_indx : a_dist == b_indx - a_indx ? a_indx : a_indx + a_dist;
    u64 d_indx = b_dist == 0 ? a_indx : b_dist == a_indx - b_indx ? b_indx : b_indx + b_dist;
    net[c_indx] = inc_port(net[c_indx], a_slot, b_indx - a_indx);
    net[d_indx] = inc_port(net[d_indx], b_slot, a_indx - b_indx);
  }
}

__host__ __device__
void chill(u64 *net, u64 len) {
  for (u64 i = 0; i < len - 1; i += 2) {
    if (get_force(net[i]) > get_force(net[i + 1])) {
      move(net, len, i, i + 1);
    }
  }
  for (u64 i = 1; i < len - 1; i += 2) {
    if (get_force(net[i]) > get_force(net[i + 1])) {
      move(net, len, i, i + 1);
    }
  }
}

__host__ __device__
bool is_valid(u64 *net, u64 len) {
  for (u64 a_indx = 0; a_indx < len; ++a_indx) {
    for (u64 a_slot = 0; a_slot < 3; ++a_slot) {
      u64 a_node = net[a_indx];
      u64 b_indx = get_dist(a_node, a_slot) + a_indx;
      u64 b_slot = get_slot(a_node, a_slot);
      u64 b_node = net[b_indx];
      if (get_dist(b_node,b_slot) != a_indx - b_indx || get_slot(b_node,b_slot) != a_slot) {
        return false;
      }
    }
  }
  return true;
}

std::vector<u64> redexes(u64 *net, u64 len) {
  std::vector<u64> redexes;
  for (u64 a_indx = 0; a_indx < len; ++a_indx) {
    u64 b_indx = get_dist(net[a_indx], 0) + a_indx;
    if (a_indx <= b_indx && redex_type(net, len, a_indx) > 0) {
      redexes.push_back(a_indx);
    }
  }
  return redexes;
}

u64 reduce_pass(u64 *net, u64 len) {
  std::vector<u64> rdx = redexes(net, len);
  u64 rwt = 0;
  for (u64 i = 0; i < rdx.size(); ++i) {
    if (rewrite(net, len, rdx[i]))  {
      rwt += 1;
    }
  }
  return rwt;
}

// ::::::::::
// :: Misc ::
// ::::::::::

std::string show_slot(u64 node, u64 slot) {
  std::string str;
  str.append(std::to_string(get_dist(node, slot)));
  switch (get_slot(node, slot)) {
    case 0: str.append("a"); break;
    case 1: str.append("b"); break;
    case 2: str.append("c"); break;
  }
  return str;
}

std::string show_node(u64 node) {
  std::string str;
  if (eql(node, air)) {
    str.append("~");
  } else if (get_type(node) == 2) {
    str.append("-(");
    str.append(show_slot(node, 1));
    str.append(" ");
    str.append(show_slot(node, 2));
    str.append(")-");
  } else {
    str.append(std::to_string(get_kind(node)));
    for (int slot = 0; slot < 3; ++slot) {
      str.append(slot > 0 ? " " : "[");
      str.append(show_slot(node, slot));
    }
    str.append("] {");
    str.append(std::to_string(get_force(node)));
    str.append("}");
    if (get_type(node) == 1) {
      str.append(" *");
    }
  }
  return str;
}

std::string plot_nums(std::vector<f64> &nums, std::vector<u64> &cols) {
  std::string str;
  for (uint i = 0; i < nums.size(); ++i) {
    str.append(cols[i] == 0 ? "\x1b[33m" : cols[i] == 1 ? "\x1b[32m" : "\x1b[31m");
    switch ((u64)(floor(fmax(fmin(nums[i],(f64)1),(f64)0) * 8))) {
      case 0: str.append(","); break;
      case 1: str.append("▁"); break;
      case 2: str.append("▂"); break;
      case 3: str.append("▃"); break;
      case 4: str.append("▄"); break;
      case 5: str.append("▅"); break;
      case 6: str.append("▆"); break;
      case 7: str.append("▇"); break;
      case 8: str.append("█"); break;
    }
    str.append("\x1b[0m");
    if (i % 128 == 127 && i < nums.size() - 1) {
      str.append("\n");
    }
  }
  return str;
};

void print_net(u64 *net, u64 len, bool show_nodes, bool show_stats, bool show_heatmap) {
  for (u64 i = 0; i < len; ++i) {
    if (show_nodes && !eql(net[i], air)) {
      std::cout << i << " - " << show_node(net[i]) << std::endl;
    }
  }
  if (show_stats) {
    std::cout << "Valid: " << is_valid(net, len) << std::endl;
  }
  if (show_heatmap) {
    std::vector<f64> nums;
    std::vector<u64> cols;
    for (u64 i = 0; i < len; ++i) {
      nums.push_back(eql(net[i], air) ? 0 : 1.0 / 8.0 + sqrt(abs(get_force(net[i]))) / 64.0);
      cols.push_back(redex_type(net, len, i));
    }
    std::cout << plot_nums(nums, cols) << std::endl;
  }
}

void print_nums(u64 *vec, u64 len) {
  for (u64 i = 0; i < len; ++i) {
    std::cout << vec[i] << " ";
  }
  std::cout << std::endl;
}

/*
// Probably not necessary, performing expansions inside shared memory instead

__global__
void expand(u64 *src, u64 *dst) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  u64 node = src[i];
  dst[i * 2 + 0] = new_node(get_kind(node),
    get_dist(node, 0) * 2, get_slot(node, 0),
    get_dist(node, 1) * 2, get_slot(node, 1),
    get_dist(node, 2) * 2, get_slot(node, 2));
  dst[i * 2 + 1] = air;
}

__global__
void shrink(u64 *src, u64 *dst, u64 *mov) {
  u64 src_indx = blockIdx.x * blockDim.x + threadIdx.x;
  u64 dst_indx = mov[src_indx];
  u64 node = src[src_indx]; 
  if (!eql(node, air)) {
    u64 x_dst_indx = mov[get_dist(node, 0) + src_indx];
    u64 y_dst_indx = mov[get_dist(node, 1) + src_indx];
    u64 z_dst_indx = mov[get_dist(node, 2) + src_indx];
    dst[dst_indx] = new_node(get_kind(node),
      (i64)x_dst_indx - (i64)dst_indx, get_slot(node, 0),
      (i64)y_dst_indx - (i64)dst_indx, get_slot(node, 1),
      (i64)z_dst_indx - (i64)dst_indx, get_slot(node, 2));
  }
}
*/

// Adapted from: https://www.mimuw.edu.pl/~ps209291/kgkp/slides/scan.pdf
// Note: maximum length = threads_per_block * 2
__device__ void scansum(u64 *data, u64 len) {
  u64 thid = threadIdx.x;
  u64 offset = 1;
  for (u64 d = len>>1; d > 0; d >>= 1) { // build sum in place up the tree
    __syncthreads();
    if (thid < d) {
      u64 ai = offset*(2*thid+1)-1;
      u64 bi = offset*(2*thid+2)-1;
      data[bi] += data[ai];
    }
    offset *= 2;
  }
  if (thid == 0) {
    data[len - 1] = 0;
  }
  for (u64 d = 1; d < len; d *= 2) { // traverse down tree & build scan
    offset >>= 1;
    __syncthreads();
    if (thid < d) {
      u64 ai = offset*(2*thid+1)-1;
      u64 bi = offset*(2*thid+2)-1;
      u64 t = data[ai];
      data[ai] = data[bi];
      data[bi] += t;
    }
  }
}

// Reduces a slice of 2048 nodes as much as possible on shared memory in parallel.
__global__ void reduce_slice(u64 *global_mem) {
  __shared__ u64 shared_mem[4096];
  __shared__ u64 new_indx[2048];
  u64 thread_mem[2];
  u64 ti = threadIdx.x;
  u64 gi = blockIdx.x * blockDim.x + ti;

  // 1. Loads a slice of 2048 nodes (16kb) from global memory to the shared memory.
  //    (There are 1024 threads. Each one is responsible for 2 nodes (128 bits).)
  for (u64 n = 0; n < 2; ++n) {
    shared_mem[ti * 2 + n] = thread_mem[n] = global_mem[gi * 2 + n];
  }

  __syncthreads();
  
  // 2. Allocates the space required for duplications by moving nodes.
  // 2a. Computes the destination index of each node with a scansum of their
  //     space needs (inactive nodes = 1, ani nodes = 0, dup nodes = 3);
  for (u64 n = 0; n < 2; ++n) {  
    u64 a_indx = ti * 2 + n;
    u64 a_node = thread_mem[n];
    u64 b_indx = get_dist(a_node, 0) + a_indx;
    u64 b_node = shared_mem[b_indx];
    new_indx[a_indx] = eql(a_node, air) ? 0 : get_redex_type(a_node, b_node) == 2 ? 3 : 1;
  }
  scansum(new_indx, 2048);
  __syncthreads();
  for (u64 n = 0; n < 2; ++n) {
    u64 a_indx = ti * 2 + n;
    shared_mem[a_indx] = air;
  }
  __syncthreads();
  for (u64 n = 0; n < 2; ++n) {
    u64 a_indx = ti * 2 + n;
    u64 a_node = thread_mem[n];
    u64 t_indx = new_indx[a_indx];
    if (!eql(a_node, air)) {
      u64 x_t_indx = new_indx[get_dist(a_node, 0) + a_indx];
      u64 y_t_indx = new_indx[get_dist(a_node, 1) + a_indx];
      u64 z_t_indx = new_indx[get_dist(a_node, 2) + a_indx];
      shared_mem[t_indx] = new_node(get_kind(a_node),
        (i64)x_t_indx - (i64)t_indx, get_slot(a_node, 0),
        (i64)y_t_indx - (i64)t_indx, get_slot(a_node, 1),
        (i64)z_t_indx - (i64)t_indx, get_slot(a_node, 2));
    }
  }

  // 3. Rewrites reducible nodes.
  //    TODO: is it possible to perform atomic links?
  for (u64 n = 0; n < 4; ++n) {
    u64 a_indx = ti * 4 + n;
    u64 a_node = thread_mem[n];
    u64 b_indx = get_dist(a_node, 0) + a_indx;
    if (b_indx < 2048) {
      u64 b_node = shared_mem[b_indx];
      // ...
    }
  }

  // 4. Movement rules.
  // TODO

  // 5. Writes results back to global memory
  for (u64 n = 0; n < 2; ++n) {
    global_mem[gi * 2 + n] = shared_mem[ti * 2 + n];
  }
}

// ::::::::::
// :: Main ::
// ::::::::::

struct is_node : public thrust::unary_function<u64,u64> {
  __host__ __device__ u64 operator()(u64 node) { return eql(node, air) ? 0 : 1; }
};

const std::vector<u64> ex = {0x0028000a00f08000,0x0028001200b8803b,0x0028001a006c7fff,0x0008001a00207fff,0x0007fff200108001,0x0027fff200088001,0x0027fff600048003,0x0017ffe5fffd8001,0x0017ffc9fffc8002,0x0027ffd600068001,0x0027ffe5fffe7fff,0x0008001a00217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0008001a00217fe5,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0017fd26000e8001,0x00280012002a7fff,0x00280012001c7fff,0x00080015fff47fff,0x0007fff200108001,0x0027fff6000c8001,0x0027fff600068001,0x00280015fffe7fff,0x0017ffc5fff47fff,0x0017ff9600008001,0x0027fffa00018000,0x0017ff6a00048001,0x0027fff600017fff,0x0027fc5200057fc4,0x0017fffa00048001,0x0027fff600017fff};  

int main(void) {
  // Creates net on host
  thrust::host_vector<u64> h_net(4096);
  thrust::fill(h_net.begin(), h_net.begin() + h_net.size(), air);
  for (int i = 0; i < ex.size(); ++i) h_net[i] = ex[i];

  reduce_pass(&h_net[0], h_net.size());
  reduce_pass(&h_net[0], h_net.size());
  reduce_pass(&h_net[0], h_net.size());

  print_net(&h_net[0], h_net.size(), true, true, true);

  // Sends to GPU
  thrust::device_vector<u64> d_net = h_net;

  // Reduces
  reduce_slice<<<1,1024>>>(thrust::raw_pointer_cast(&d_net[0]));
  
  // Sends to CPU & prints
  h_net = d_net;
  print_net(&h_net[0], h_net.size(), true, true, true);
  /*print_nums(&h_net[0], h_net.size());*/


  return 0;
}

