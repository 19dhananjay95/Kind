#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <string>
#include <iostream>
#include <vector>

typedef  int64_t i64;
typedef uint64_t u64;
typedef double   f64;

// ::::::::::
// :: Node ::
// ::::::::::

u64 new_node(u64 kind, i64 a_dist, u64 a_slot, i64 b_dist, u64 b_slot, i64 c_dist, u64 c_slot) {
  return (kind << 54)
      | (a_slot << 52) | ((u64)(a_dist + 32768) << 36)
      | (b_slot << 34) | ((u64)(b_dist + 32768) << 18)
      | (c_slot << 16) | ((u64)(c_dist + 32768) <<  0);
}

u64 get_kind(u64 node) {
  return (node >> 54) & 0xFF;
}

i64 get_dist(u64 node, u64 slot) {
  return (i64)((node >> (36 - slot * 18)) & 0xFFFF) - 32768;
}

u64 get_slot(u64 node, u64 slot) {
  return ((node >> (52 - slot * 18))  & 0x3);
}

u64 inc_port(u64 node, u64 slot, i64 delta) {
  return (u64)((i64)node + (delta << (36 - slot * 18)));
}

u64 mov_node(u64 node, i64 delta) {
  return (u64)((i64)node + (-delta << 36) + (-delta << 18) + -delta);
}

u64 set_port(u64 node, u64 slot, i64 new_dist, u64 new_slot) {
  return node & ~((u64)0x3FFFF << (36 - slot * 18)) | (((new_slot << 16) | (u64)(new_dist + 32768)) << (36 - slot * 18));
}

u64 eql(u64 a, u64 b) {
  return a == b;
}

f64 get_force(u64 node) {
  i64 x = get_dist(node, 0);
  i64 y = get_dist(node, 1);
  i64 z = get_dist(node, 2);
  return (f64)((x < 0 ? -1 : 1) * x * x + (y < 0 ? -1 : 1) * y * y + (z < 0 ? -1 : 1) * z * z);
}

const u64 air = new_node(0, 0,0, 0,1, 0,2);

// :::::::::
// :: Net ::
// :::::::::

typedef std::vector<u64> Net;

u64 alloc(Net &net) {
  net.push_back(air);
  return net.size() - 1;
}

void link(Net &net, u64 a_indx, u64 a_slot, u64 b_indx, u64 b_slot) {
  net[a_indx] = set_port(net[a_indx], a_slot, b_indx - a_indx, b_slot);
  net[b_indx] = set_port(net[b_indx], b_slot, a_indx - b_indx, a_slot);
}

void unlink(Net &net, u64 a_indx, u64 a_slot) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, a_slot) + a_indx;
  u64 b_slot = get_slot(a_node, a_slot);
  u64 b_node = net[b_indx];
  if (get_dist(b_node, b_slot) + b_indx == a_indx && get_slot(b_node, b_slot) == a_slot) {
    net[a_indx] = set_port(a_node, a_slot, 0, a_slot);
    net[b_indx] = set_port(b_node, b_slot, 0, b_slot);
  }
}

bool is_redex(const Net &net, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, 0) + a_indx;
  u64 b_node = net[b_indx];
  return get_slot(a_node, 0) == 0 && (get_dist(a_node, 0) + get_dist(b_node, 0)) == 0 && !eql(a_node, air);
};

void rewrite(Net &net, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = a_indx + get_dist(a_node, 0);
  u64 b_node = net[b_indx];
  if (get_kind(a_node) == get_kind(b_node)) {
    u64 a1_indx = get_dist(net[a_indx], 1) + a_indx;
    u64 a1_slot = get_slot(net[a_indx], 1);
    u64 b1_indx = get_dist(net[b_indx], 1) + b_indx;
    u64 b1_slot = get_slot(net[b_indx], 1);
    link(net, a1_indx, a1_slot, b1_indx, b1_slot);
    u64 a2_indx = get_dist(net[a_indx], 2) + a_indx;
    u64 a2_slot = get_slot(net[a_indx], 2);
    u64 b2_indx = get_dist(net[b_indx], 2) + b_indx;
    u64 b2_slot = get_slot(net[b_indx], 2);
    link(net, a2_indx, a2_slot, b2_indx, b2_slot);
  } else {
    u64 c_indx = alloc(net);
    u64 d_indx = alloc(net);
    u64 e_indx = alloc(net);
    u64 f_indx = alloc(net);
    net[c_indx] = new_node(get_kind(b_node), 0,0, f_indx - c_indx, 1, e_indx - c_indx, 1); 
    net[d_indx] = new_node(get_kind(b_node), 0,0, f_indx - d_indx, 2, e_indx - d_indx, 2); 
    net[e_indx] = new_node(get_kind(a_node), 0,0, c_indx - e_indx, 2, d_indx - e_indx, 2);
    net[f_indx] = new_node(get_kind(a_node), 0,0, c_indx - f_indx, 1, d_indx - f_indx, 1);
    link(net, c_indx, 0, get_dist(net[a_indx],1) + a_indx, get_slot(net[a_indx],1));
    link(net, d_indx, 0, get_dist(net[a_indx],2) + a_indx, get_slot(net[a_indx],2));
    link(net, e_indx, 0, get_dist(net[b_indx],2) + b_indx, get_slot(net[b_indx],2));
    link(net, f_indx, 0, get_dist(net[b_indx],1) + b_indx, get_slot(net[b_indx],1));
  }
  for (int slot = 0; slot < 3; slot++) {
    unlink(net, a_indx, slot);
    unlink(net, b_indx, slot);
  }
  net[a_indx] = air;
  net[b_indx] = air;
}

void move(Net &net, u64 a_indx, u64 b_indx) {
  u64 a_node = net[a_indx];
  u64 b_node = net[b_indx];
  net[b_indx] = mov_node(a_node, b_indx - a_indx);
  net[a_indx] = mov_node(b_node, a_indx - b_indx);
  for (u64 slot = 0; slot < 3; ++slot) {
    u64 a_dist = get_dist(a_node, slot);
    u64 a_slot = get_slot(a_node, slot);
    u64 b_dist = get_dist(b_node, slot);
    u64 b_slot = get_slot(b_node, slot);
    u64 c_indx = a_dist == 0 ? b_indx : a_dist == b_indx - a_indx ? a_indx : a_indx + a_dist;
    u64 d_indx = b_dist == 0 ? a_indx : b_dist == a_indx - b_indx ? b_indx : b_indx + b_dist;
    net[c_indx] = inc_port(net[c_indx], a_slot, b_indx - a_indx);
    net[d_indx] = inc_port(net[d_indx], b_slot, a_indx - b_indx);
  }
}

bool is_valid(const Net &net) {
  for (u64 a_indx = 0; a_indx < net.size(); ++a_indx) {
    for (u64 a_slot = 0; a_slot < 3; ++a_slot) {
      u64 a_node = net[a_indx];
      u64 b_indx = get_dist(a_node, a_slot) + a_indx;
      u64 b_slot = get_slot(a_node, a_slot);
      u64 b_node = net[b_indx];
      if (get_dist(b_node,b_slot) != a_indx - b_indx || get_slot(b_node,b_slot) != a_slot) {
        return false;
      }
    }
  }
  return true;
}

/*
// TODO: port from JS
void reduce_pass(Net &net) => {
  u32 rdx = redexes(net);
  var rwt = 0;
  for (var i = 0; i < rdx.length; ++i) {
    if (!only_ani || kin(net[rdx[i][0]]) === kin(net[rdx[i][1]])) {
      if (rewrite(net, rdx[i][0])) {
        rwt += 1;
      }
    }
  }
  return rwt;
};
*/


// ::::::::::
// :: Misc ::
// ::::::::::

std::string show_slot(u64 node, u64 slot) {
  std::string str;
  str.append(std::to_string(get_dist(node, slot)));
  switch (get_slot(node, slot)) {
    case 0: str.append("a"); break;
    case 1: str.append("b"); break;
    case 2: str.append("c"); break;
  }
  return str;
}

std::string show_node(u64 node) {
  std::string str;
  str.append(std::to_string(get_kind(node)));
  for (int slot = 0; slot < 3; ++slot) {
    str.append(slot > 0 ? " " : "[");
    str.append(show_slot(node, slot));
  }
  str.append("] {");
  str.append(std::to_string(get_force(node)));
  str.append("}");
  return str;
}

std::string plot_nums(std::vector<f64> &nums, std::vector<bool> &mark) {
  std::string str;
  for (uint i = 0; i < nums.size(); ++i) {
    str.append(mark[i] ? "\x1b[32m" : "\x1b[31m");
    switch ((u64)(floor(max(min(nums[i],(f64)1),(f64)0) * 8))) {
      case 0: str.append(" "); break;
      case 1: str.append("▁"); break;
      case 2: str.append("▂"); break;
      case 3: str.append("▃"); break;
      case 4: str.append("▄"); break;
      case 5: str.append("▅"); break;
      case 6: str.append("▆"); break;
      case 7: str.append("▇"); break;
      case 8: str.append("█"); break;
    }
    str.append("\x1b[0m");
  }
  return str;
};

void print_net(const Net &net) {
  std::vector<f64> nums;
  std::vector<bool> mark;
  for (u64 i = 0; i < net.size(); ++i) {
    /*std::cout << i << " - " << show_node(net[i]) << std::endl;*/
    nums.push_back(1.0 / 8.0 + sqrt(abs(get_force(net[i]))) / 64.0);
    mark.push_back(is_redex(net, i));
  }
  std::cout << plot_nums(nums, mark) << std::endl;
}

// ::::::::::
// :: Main ::
// ::::::::::

const std::vector<u64> net = {0x0028000a00f08000,0x0028001200b8803b,0x0028001a006c7fff,0x0008001a00207fff,0x0007fff200108001,0x0027fff200088001,0x0027fff600048003,0x0017ffe5fffd8001,0x0017ffc9fffc8002,0x0027ffd600068001,0x0027ffe5fffe7fff,0x0008001a00217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0008001a00217fe5,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0017fd26000e8001,0x00280012002a7fff,0x00280012001c7fff,0x00080015fff47fff,0x0007fff200108001,0x0027fff6000c8001,0x0027fff600068001,0x00280015fffe7fff,0x0017ffc5fff47fff,0x0017ff9600008001,0x0027fffa00018000,0x0017ff6a00048001,0x0027fff600017fff,0x0027fc5200057fc4,0x0017fffa00048001,0x0027fff600017fff};  

int main(void) {
  /*std::cout << show_node(inc_port(set_port(air, 2, 7,0), 0, 3)) << std::endl;*/

  print_net(net);
  
  return 0;
}
