
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string>
#include <iostream>
#include <vector>

typedef  int64_t i64;
typedef uint64_t u64;

u64 new_node(u64 kind, i64 a_dist, u64 a_slot, i64 b_dist, u64 b_slot, i64 c_dist, u64 c_slot) {
  return (kind << 54)
      | (a_slot << 52) | ((u64)(a_dist + 32768) << 36)
      | (b_slot << 34) | ((u64)(b_dist + 32768) << 18)
      | (c_slot << 16) | ((u64)(c_dist + 32768) <<  0);
}

u64 get_kind(u64 node) {
  return (node >> 54) & 0xFF;
}

i64 get_dist(u64 node, u64 slot) {
  return (i64)((node >> (36 - slot * 18)) & 0xFFFF) - 32768;
}

u64 get_slot(u64 node, u64 slot) {
  return ((node >> (52 - slot * 18))  & 0x3);
}

u64 inc_port(u64 node, u64 slot, i64 delta) {
  return (u64)((i64)node + (delta << (36 - slot * 18)));
}

u64 mov_node(u64 node, i64 delta) {
  return (u64)((i64)node + (-delta << 36) + (-delta << 18) + -delta);
}

u64 set_port(u64 node, u64 slot, i64 new_dist, u64 new_slot) {
  return node & ~((u64)0x3FFFF << (36 - slot * 18)) | (((new_slot << 16) | (u64)(new_dist + 32768)) << (36 - slot * 18));
}

u64 eql(u64 a, u64 b) {
  return a == b;
}

const u64 air = new_node(0, 0,0, 0,1, 0,2);

u64 alloc(std::vector<u64> &net) {
  net.push_back(air);
  return net.size() - 1;
}

void link(std::vector<u64> &net, u64 a_indx, u64 a_slot, u64 b_indx, u64 b_slot) {
  net[a_indx] = set_port(net[a_indx], a_slot, b_indx - a_indx, b_slot);
  net[b_indx] = set_port(net[b_indx], b_slot, a_indx - b_indx, a_slot);
}

void unlink(std::vector<u64> &net, u64 a_indx, u64 a_slot) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, a_slot) + a_indx;
  u64 b_slot = get_slot(a_node, a_slot);
  u64 b_node = net[b_indx];
  if (get_dist(b_node, b_slot) + b_indx == a_indx && get_slot(b_node, b_slot) == a_slot) {
    net[a_indx] = set_port(a_node, a_slot, 0, a_slot);
    net[b_indx] = set_port(b_node, b_slot, 0, b_slot);
  }
}

std::string show_slot(u64 node, u64 slot) {
  std::string str;
  str.append(std::to_string(get_dist(node, slot)));
  switch (get_slot(node, slot)) {
    case 0: str.append("a"); break;
    case 1: str.append("b"); break;
    case 2: str.append("c"); break;
  }
  return str;
}

std::string show_node(u64 node) {
  std::string str;
  str.append(std::to_string(get_kind(node)));
  for (int slot = 0; slot < 3; ++slot) {
    str.append(slot > 0 ? " " : "[");
    str.append(show_slot(node, slot));
  }
  str.append("]");
  return str;
}


/*var nod = (k,ad,as,bd,bs,cd,cs) => (1<<31)|(k<<30)|(as<<28)|((ad+128)<<20)|(bs<<18)|((bd+128)<<10)|(cs<<8)|(cd+128);*/
/*var kin = (nod) => (nod >>> 30) & 0x1; // get kind*/
/*var dst = (nod,s) => ((nod >>> (20 - s * 10)) & 0xFF) - 128; // get target distance of port `s`*/
/*var slt = (nod,s) => ((nod >>> (28 - s * 10)) & 0x3); // get target slot of port `s`*/
/*var add = (nod,s,k) => nod + (k << (20 - s * 10)); // add k to slot s*/
/*var mov = (nod,d) => nod + (-d << 20) + (-d << 10) + -d; // dd d to all slots*/
/*var set = (nod,s,D,S) => nod&~(0x3FF<<(20-s*10))|(((S<<8)|(D+128))<<(20-s*10)); // set ptr on slot s to ti,ts*/
/*var eql = (a,b) => a === b;*/
/*var air = nod(0, 0,0, 0,1, 0,2);*/
/*var ela = (n) => Math.sign(n) * n * n;*/
/*var pow = (nod) => ela(dst(nod,0)) + ela(dst(nod,1)) + ela(dst(nod,2));*/
/*var max = (nod) => Math.max(Math.abs(dst(nod,0)), Math.max(Math.abs(dst(nod,1)), Math.abs(dst(nod,2))));*/
/*var sho = (nod) => eql(nod, air) ? "~" : kin(nod) + "[" + dst(nod,0) + "abc"[slt(nod,0)] + "|" + dst(nod,1) + "abc"[slt(nod,1)] + "|" + dst(nod,2) + "abc"[slt(nod,2)] + "] " + "{" + pow(nod).toFixed(2) + "}";*/
/*var str = net => net.map((k,i) => "| " + ("0000" + i).slice(-4) + " : " + sho(k)).join("\n");*/

int main(void) {
  std::cout << show_node(inc_port(set_port(air, 2, 7,0), 0, 3)) << std::endl;
  /*printf("hi");*/
  
  return 0;
}
