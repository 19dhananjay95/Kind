#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <string>
#include <iostream>
#include <vector>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>
#include <thrust/scan.h>

typedef  int64_t i64;
typedef uint64_t u64;
typedef double   f64;

// ::::::::::
// :: Node ::
// ::::::::::

// type=0 is a normal node
// type=1 is a duplicating node
// type=2 is a wire
__host__ __device__
u64 new_node(u64 kind, i64 a_dist, u64 a_slot, i64 b_dist, u64 b_slot, i64 c_dist, u64 c_slot) {
  return (kind << 54)
      | (a_slot << 52) | ((u64)(a_dist + 32768) << 36)
      | (b_slot << 34) | ((u64)(b_dist + 32768) << 18)
      | (c_slot << 16) | ((u64)(c_dist + 32768) <<  0);
}

__constant__
const u64 air = 0x8000600028000; // new_node(0, 0,0, 0,1, 0,2)
const u64 dot = 11259024838262784;

__host__ __device__ 
u64 to_wire(u64 node) {
  return (node & ~((u64)0x1 << 63)) | ((u64)1 << 63);
}

__host__ __device__
u64 is_wire(u64 node) {
  return (node >> 63) & 0x1;
}

__host__ __device__
u64 get_kind(u64 node) {
  return (node >> 54) & 0xFF;
}

__host__ __device__
i64 get_dist(u64 node, u64 slot) {
  return (i64)((node >> (36 - slot * 18)) & 0xFFFF) - 32768;
}

__host__ __device__
u64 get_slot(u64 node, u64 slot) {
  return ((node >> (52 - slot * 18)) & 0x3);
}

__host__ __device__
u64 inc_port(u64 node, u64 slot, i64 delta) {
  return (u64)((i64)node + (delta << (36 - slot * 18)));
}

__host__ __device__
u64 inc_ports(u64 node, i64 delta) {
  return (u64)((i64)node + (delta << 36) + (delta << 18) + delta);
}

__host__ __device__
u64 set_port(u64 node, u64 slot, i64 new_dist, u64 new_slot) {
  return (node & ~((u64)0x3FFFF << (36 - slot * 18))) | (((new_slot << 16) | (u64)(new_dist + 32768)) << (36 - slot * 18));
}

__host__ __device__
u64 eql(u64 a, u64 b) {
  return a == b;
}

__host__ __device__
f64 get_force(u64 node) {
  i64 x = get_dist(node, 0);
  i64 y = get_dist(node, 1);
  i64 z = get_dist(node, 2);
  return (f64)((x < 0 ? -1 : 1) * x * x + (y < 0 ? -1 : 1) * y * y + (z < 0 ? -1 : 1) * z * z);
}

__host__ __device__
u64 get_redex_type(u64 a_node, u64 b_node) {
  if (!eql(a_node, air) && !eql(b_node, air) && get_dist(a_node, 0) + get_dist(b_node, 0) == 0 && get_slot(a_node, 0) == 0) {
    return get_kind(a_node) == get_kind(b_node) ? 1 : 2;
  } else {
    return 0;
  }
}

// :::::::::
// :: Net ::
// :::::::::

struct Alloc {
  u64 indxs[4];
};

__host__ __device__
bool alloc4(u64 *net, u64 len, u64 i, u64 *indxs) {
  u64 k = 0, n, a;
  u64 j = 0;
  do {
    k = k + 1;
    n = i + ((k % 2) * 2 - 1) * (k / 2);
    a = n < len ? net[n] : 0;
    if (eql(a, air)) {
      indxs[j++] = n;
    }
  } while (k < 32 && j < 4);
  return j == 4;
}

__host__ __device__
void link(u64* net, u64 len, u64 a_indx, u64 a_slot, u64 b_indx, u64 b_slot) {
  net[a_indx] = set_port(net[a_indx], a_slot, b_indx - a_indx, b_slot);
  net[b_indx] = set_port(net[b_indx], b_slot, a_indx - b_indx, a_slot);
}

__host__ __device__
void unlink(u64 *net, u64 len, u64 a_indx, u64 a_slot) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, a_slot) + a_indx;
  u64 b_slot = get_slot(a_node, a_slot);
  u64 b_node = net[b_indx];
  if (get_dist(b_node, b_slot) + b_indx == a_indx && get_slot(b_node, b_slot) == a_slot) {
    net[a_indx] = set_port(a_node, a_slot, 0, a_slot);
    net[b_indx] = set_port(b_node, b_slot, 0, b_slot);
  }
}

__host__ __device__
u64 get_redex_type_at(u64* net, u64 len, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = get_dist(a_node, 0) + a_indx;
  u64 b_node = net[b_indx];
  return get_redex_type(a_node, b_node);
};

__host__ __device__
bool rewrite(u64* net, u64 len, u64 a_indx) {
  u64 a_node = net[a_indx];
  u64 b_indx = a_indx + get_dist(a_node, 0);
  u64 b_node = net[b_indx];
  if (get_redex_type_at(net, len, a_indx) == 0) return false;
  if (get_kind(a_node) == get_kind(b_node)) {
    u64 a1_indx = get_dist(net[a_indx], 1) + a_indx;
    u64 a1_slot = get_slot(net[a_indx], 1);
    u64 b1_indx = get_dist(net[b_indx], 1) + b_indx;
    u64 b1_slot = get_slot(net[b_indx], 1);
    link(net, len, a1_indx, a1_slot, b1_indx, b1_slot);
    u64 a2_indx = get_dist(net[a_indx], 2) + a_indx;
    u64 a2_slot = get_slot(net[a_indx], 2);
    u64 b2_indx = get_dist(net[b_indx], 2) + b_indx;
    u64 b2_slot = get_slot(net[b_indx], 2);
    link(net, len, a2_indx, a2_slot, b2_indx, b2_slot);
  } else {
    u64 indxs[4] = {0, 0, 0, 0};
    if (!alloc4(net, len, (a_indx + b_indx) / 2, indxs)) return false;
    u64 c_indx = indxs[0];
    u64 d_indx = indxs[1];
    u64 e_indx = indxs[2];
    u64 f_indx = indxs[3];
    net[c_indx] = new_node(get_kind(b_node), 0,0, f_indx - c_indx, 1, e_indx - c_indx, 1); 
    net[d_indx] = new_node(get_kind(b_node), 0,0, f_indx - d_indx, 2, e_indx - d_indx, 2); 
    net[e_indx] = new_node(get_kind(a_node), 0,0, c_indx - e_indx, 2, d_indx - e_indx, 2);
    net[f_indx] = new_node(get_kind(a_node), 0,0, c_indx - f_indx, 1, d_indx - f_indx, 1);
    link(net, len, c_indx, 0, get_dist(net[a_indx],1) + a_indx, get_slot(net[a_indx],1));
    link(net, len, d_indx, 0, get_dist(net[a_indx],2) + a_indx, get_slot(net[a_indx],2));
    link(net, len, e_indx, 0, get_dist(net[b_indx],2) + b_indx, get_slot(net[b_indx],2));
    link(net, len, f_indx, 0, get_dist(net[b_indx],1) + b_indx, get_slot(net[b_indx],1));
  }
  for (int slot = 0; slot < 3; slot++) {
    unlink(net, len, a_indx, slot);
    unlink(net, len, b_indx, slot);
  }
  net[a_indx] = air;
  net[b_indx] = air;
  return true;
}

__host__ __device__
void move(u64 *net, u64 len, u64 a_indx, u64 b_indx) {
  u64 a_node = net[a_indx];
  u64 b_node = net[b_indx];
  net[b_indx] = inc_ports(a_node, -(b_indx - a_indx));
  net[a_indx] = inc_ports(b_node, -(a_indx - b_indx));
  for (u64 slot = 0; slot < 3; ++slot) {
    u64 a_dist = get_dist(a_node, slot);
    u64 a_slot = get_slot(a_node, slot);
    u64 b_dist = get_dist(b_node, slot);
    u64 b_slot = get_slot(b_node, slot);
    u64 c_indx = a_dist == 0 ? b_indx : a_dist == b_indx - a_indx ? a_indx : a_indx + a_dist;
    u64 d_indx = b_dist == 0 ? a_indx : b_dist == a_indx - b_indx ? b_indx : b_indx + b_dist;
    net[c_indx] = inc_port(net[c_indx], a_slot, b_indx - a_indx);
    net[d_indx] = inc_port(net[d_indx], b_slot, a_indx - b_indx);
  }
}

__host__ __device__
void chill(u64 *net, u64 len) {
  for (u64 i = 0; i < len - 1; i += 2) {
    if (get_force(net[i]) > get_force(net[i + 1])) {
      move(net, len, i, i + 1);
    }
  }
  for (u64 i = 1; i < len - 1; i += 2) {
    if (get_force(net[i]) > get_force(net[i + 1])) {
      move(net, len, i, i + 1);
    }
  }
}

bool is_valid(u64 *net, u64 len) {
  u64 a_indx, a_slot, a_node;
  u64 b_indx, b_slot, b_node; 
  u64 c_indx, c_slot, c_node; 
  for (a_indx = 0; a_indx < len; ++a_indx) {
    a_node = net[a_indx];
    if (!eql(a_node, air) && !is_wire(a_node)) {
      for (a_slot = 0; a_slot < 3; ++a_slot) {
        b_indx = get_dist(a_node, a_slot) + a_indx;
        b_slot = get_slot(a_node, a_slot);
        b_node = net[b_indx];
        while (is_wire(b_node)) {
          b_indx = get_dist(b_node, b_slot) + b_indx;
          b_slot = get_slot(b_node, b_slot);
          b_node = net[b_indx];
        }
        c_indx = get_dist(b_node, b_slot) + b_indx;
        c_slot = get_slot(b_node, b_slot);
        c_node = net[c_indx];
        while (is_wire(c_node)) {
          c_indx = get_dist(c_node, c_slot) + c_indx;
          c_slot = get_slot(c_node, c_slot);
          c_node = net[c_indx];
        }
        if (a_indx != c_indx || a_slot != c_slot) {
          std::cout << "bad " << a_indx << ":" << a_slot << " " << b_indx << ":" << b_slot << " " << c_indx << ":" << b_slot << std::endl;
          return false;
        }
      }
    }
  }
  return true;
}

std::vector<u64> redexes(u64 *net, u64 len) {
  std::vector<u64> redexes;
  for (u64 a_indx = 0; a_indx < len; ++a_indx) {
    u64 b_indx = get_dist(net[a_indx], 0) + a_indx;
    if (a_indx <= b_indx && get_redex_type_at(net, len, a_indx) > 0) {
      redexes.push_back(a_indx);
    }
  }
  return redexes;
}

u64 reduce_pass(u64 *net, u64 len) {
  std::vector<u64> rdx = redexes(net, len);
  u64 rwt = 0;
  for (u64 i = 0; i < rdx.size(); ++i) {
    if (rewrite(net, len, rdx[i]))  {
      rwt += 1;
    }
  }
  return rwt;
}

// ::::::::::
// :: Misc ::
// ::::::::::

std::string show_slot(u64 node, u64 slot) {
  std::string str;
  str.append(std::to_string(get_dist(node, slot)));
  switch (get_slot(node, slot)) {
    case 0: str.append("a"); break;
    case 1: str.append("b"); break;
    case 2: str.append("c"); break;
  }
  return str;
}

std::string show_node(u64 node) {
  std::string str;
  if (eql(node, air)) {
    str.append("~");
  } else {
    if (is_wire(node)) {
      str.append("-");
    } else {
      str.append(std::to_string(get_kind(node)));
    }
    for (int slot = 0; slot < 3; ++slot) {
      str.append(slot > 0 ? " " : "[");
      str.append(show_slot(node, slot));
    }
    str.append("] {");
    str.append(std::to_string(get_force(node)));
    str.append("}");
  }
  return str;
}

std::string plot_nums(std::vector<f64> &nums, std::vector<u64> &cols) {
  std::string str;
  for (uint i = 0; i < nums.size(); ++i) {
    str.append(cols[i] == 0 ? "\x1b[33m" : cols[i] == 1 ? "\x1b[32m" : cols[i] == 2 ? "\x1b[31m" : cols[i] == 3 ? "\x1b[34m" : "\x1b[35m");
    switch ((u64)(floor(fmax(fmin(nums[i],(f64)1),(f64)0) * 8))) {
      case 0: str.append(","); break;
      case 1: str.append("▁"); break;
      case 2: str.append("▂"); break;
      case 3: str.append("▃"); break;
      case 4: str.append("▄"); break;
      case 5: str.append("▅"); break;
      case 6: str.append("▆"); break;
      case 7: str.append("▇"); break;
      case 8: str.append("█"); break;
    }
    str.append("\x1b[0m");
    if (i % 128 == 127 && i < nums.size() - 1) {
      str.append("\n");
    }
  }
  return str;
};

void print_net(u64 *net, u64 len, bool show_nodes, bool show_stats, bool show_heatmap) {
  for (u64 i = 0; i < len; ++i) {
    if (show_nodes && !eql(net[i], air)) {
      std::cout << i << " - " << show_node(net[i]) << std::endl;
    }
  }
  if (show_stats) {
    std::cout << "Valid: " << is_valid(net, len) << std::endl;
  }
  if (show_heatmap) {
    std::vector<f64> nums;
    std::vector<u64> cols;
    for (u64 i = 0; i < len; ++i) {
      nums.push_back(eql(net[i], air) ? 0 : 1.0 / 8.0 + sqrt(abs(get_force(net[i]))) / 64.0);
      cols.push_back(is_wire(net[i]) ? 3 : eql(net[i],dot) ? 4 : get_redex_type_at(net, len, i));
    }
    std::cout << plot_nums(nums, cols) << std::endl;
  }
}

void print_nums(u64 *vec, u64 len) {
  for (u64 i = 0; i < len; ++i) {
    std::cout << vec[i] << " ";
  }
  std::cout << std::endl;
}

/*
// Probably not necessary, performing expansions inside shared memory instead

__global__
void expand(u64 *src, u64 *dst) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  u64 node = src[i];
  dst[i * 2 + 0] = new_node(get_kind(node),
    get_dist(node, 0) * 2, get_slot(node, 0),
    get_dist(node, 1) * 2, get_slot(node, 1),
    get_dist(node, 2) * 2, get_slot(node, 2));
  dst[i * 2 + 1] = air;
}

__global__
void shrink(u64 *src, u64 *dst, u64 *mov) {
  u64 src_indx = blockIdx.x * blockDim.x + threadIdx.x;
  u64 dst_indx = mov[src_indx];
  u64 node = src[src_indx]; 
  if (!eql(node, air)) {
    u64 x_dst_indx = mov[get_dist(node, 0) + src_indx];
    u64 y_dst_indx = mov[get_dist(node, 1) + src_indx];
    u64 z_dst_indx = mov[get_dist(node, 2) + src_indx];
    dst[dst_indx] = new_node(get_kind(node),
      (i64)x_dst_indx - (i64)dst_indx, get_slot(node, 0),
      (i64)y_dst_indx - (i64)dst_indx, get_slot(node, 1),
      (i64)z_dst_indx - (i64)dst_indx, get_slot(node, 2));
  }
}
*/

// Adapted from: https://www.mimuw.edu.pl/~ps209291/kgkp/slides/scan.pdf
// Note: maximum length = threads_per_block * 2
__device__ void scansum(u64 *data, u64 len) {
  u64 thid = threadIdx.x;
  u64 offset = 1;
  for (u64 d = len>>1; d > 0; d >>= 1) { // build sum in place up the tree
    __syncthreads();
    if (thid < d) {
      u64 ai = offset*(2*thid+1)-1;
      u64 bi = offset*(2*thid+2)-1;
      data[bi] += data[ai];
    }
    offset *= 2;
  }
  if (thid == 0) {
    data[len - 1] = 0;
  }
  for (u64 d = 1; d < len; d *= 2) { // traverse down tree & build scan
    offset >>= 1;
    __syncthreads();
    if (thid < d) {
      u64 ai = offset*(2*thid+1)-1;
      u64 bi = offset*(2*thid+2)-1;
      u64 t = data[ai];
      data[ai] = data[bi];
      data[bi] += t;
    }
  }
}

// Reduces a block of 2048 nodes on shared memory.
// `snet` is the block with the input nodes that will be reduced.
// `spos` is the global position of input nodes, which will be updated.
__device__ void reduce_block(u64 *snet, u64 *spos) {
  __shared__ u64 mov2[2048];
  u64 tmem[6];
  u64 a_indx, a_slot, a_node;
  u64 b_indx, b_slot, b_node;
  u64 rdx_ty, n, t_indx[3];
  u64 ti = threadIdx.x;

  // Computes the dest index of each node by scan-summing their space neds.
  for (n = 0; n < 2; ++n) { 
    a_indx = ti * 2 + n;
    a_node = snet[a_indx];
    b_indx = get_dist(a_node, 0) + a_indx;
    b_node = snet[b_indx];
    rdx_ty = get_redex_type(a_node, b_node);
    mov2[a_indx] = eql(a_node, air) ? 0 : rdx_ty == 2 ? 3 : 1;
  }
  scansum(mov2, 2048);
  __syncthreads();

  // Moves net to their target locations.
  for (n = 0; n < 2; ++n) {
    a_indx = ti * 2 + n;
    tmem[n] = snet[a_indx];
  }
  __syncthreads();
  for (n = 0; n < 2; ++n) {
    a_indx = ti * 2 + n;
    snet[a_indx] = air;
  }
  __syncthreads();
  for (n = 0; n < 2; ++n) {
    a_indx = ti * 2 + n;
    a_node = tmem[n];
    b_indx = mov2[a_indx];
    if (!eql(a_node, air)) {
      for (b_slot = 0; b_slot < 3; ++b_slot) {
        t_indx[b_slot] = get_dist(a_node, b_slot) + a_indx;
        t_indx[b_slot] = t_indx[b_slot] < 2048 ? mov2[t_indx[b_slot]] : t_indx[b_slot];
      }
      snet[b_indx] = new_node(get_kind(a_node),
        (i64)t_indx[0] - (i64)b_indx, get_slot(a_node, 0),
        (i64)t_indx[1] - (i64)b_indx, get_slot(a_node, 1),
        (i64)t_indx[2] - (i64)b_indx, get_slot(a_node, 2));
    /*} else {*/
      /*mov2[a_indx] = 0xFFFFFFFF;*/
    }
  }
  __syncthreads();

  /*// Performs annihilation and duplication reductions.*/
  for (n = 0; n < 2; ++n) {
    a_indx = ti * 2 + n;
    a_node = snet[a_indx];
    tmem[n * 3 + 0] = a_node;
    tmem[n * 3 + 1] = air;
    tmem[n * 3 + 2] = air;
    if (!eql(a_node, air)) {
      b_indx = get_dist(a_node, 0) + a_indx;
      b_node = snet[b_indx];
      rdx_ty = get_redex_type(a_node, b_node);
      if (rdx_ty == 2) {
        tmem[n * 3 + 0] = to_wire(new_node(0, 0, 0, 1, 0, 2, 0));
        tmem[n * 3 + 1] = new_node(get_kind(b_node), get_dist(a_node, 1) - 1, get_slot(a_node, 1), (i64)(b_indx+1) - (i64)(a_indx+1), 1, (i64)(b_indx+2) - (i64)(a_indx+1), 1);
        tmem[n * 3 + 2] = new_node(get_kind(b_node), get_dist(a_node, 2) - 2, get_slot(a_node, 2), (i64)(b_indx+1) - (i64)(a_indx+2), 2, (i64)(b_indx+2) - (i64)(a_indx+2), 2);
      }
      if (rdx_ty == 1) {
        tmem[n * 3 + 0] = to_wire(new_node(0, 0, 0, (i64)b_indx - (i64)a_indx + get_dist(b_node, 1), get_slot(b_node, 1), (i64)b_indx - (i64)a_indx + get_dist(b_node, 2), get_slot(b_node, 2)));
      }
    }
  }
  __syncthreads();
  for (n = 0; n < 2; ++n) {
    a_indx = ti * 2 + n;
    a_node = tmem[n * 3 + 0];
    if (!eql(a_node, air)) {
      snet[a_indx + 0] = tmem[n * 3 + 0];
      if (!eql(tmem[n * 3 + 1], air)) {
        snet[a_indx + 1] = tmem[n * 3 + 1];
        snet[a_indx + 2] = tmem[n * 3 + 2];
      }
    }
  }
  __syncthreads();

  /*// Connects linked ports and cleans unused space.*/
  for (n = 0; n < 2; ++n) {
    a_indx = ti * 2 + n;
    a_node = snet[a_indx];
    if (is_wire(a_node)) {
      tmem[n] = air;
    } else {
      tmem[n] = a_node;
      if (!eql(a_node, air)) {
        for (a_slot = 0; a_slot < 3; ++a_slot) {
          b_indx = get_dist(a_node, a_slot) + a_indx;
          b_slot = get_slot(a_node, a_slot);
          b_node = snet[b_indx];
          while (is_wire(b_node)) {
            b_indx = get_dist(b_node, b_slot) + b_indx;
            b_slot = get_slot(b_node, b_slot);
            b_node = snet[b_indx];
          }
          tmem[n] = set_port(tmem[n], a_slot, b_indx - a_indx, b_slot);
        }
      }
    }
  }
  __syncthreads();
  for (n = 0; n < 2; ++n) {
    snet[ti*2+n] = tmem[n];
    if (spos[ti*2+n] != 0xFFFFFFFF) {
      spos[ti*2+n] = mov2[spos[ti*2+n]];
    }
  }
  __syncthreads();
}

// Reduces all blocks in parallel.
__global__ void reduce_blocks(u64 *gnet, u64 *gpos) {
  __shared__ u64 snet[2048];
  __shared__ u64 spos[2048];

  u64 ti = threadIdx.x;
  u64 gi = blockIdx.x * blockDim.x + ti;

  // Loads a block of 2048 nodes (16kb) from global memory.
  for (u64 n = 0; n < 2; ++n) {
    snet[ti*2+n] = gnet[gi*2+n];
    spos[ti*2+n] = gpos[gi*2+n];
  }
  __syncthreads();

  // Performs a few parallel reductions.
  for (u64 n = 0; n < 256; ++n) {
    reduce_block(snet, spos);
  }
  
  // Writes results back to global memory
  for (u64 n = 0; n < 2; ++n) {
    gnet[gi*2+n] = snet[ti*2+n];
    gpos[gi*2+n] = spos[ti*2+n];
  }
}

// ::::::::::
// :: Main ::
// ::::::::::

struct is_node : public thrust::unary_function<u64,u64> {
  __host__ __device__ u64 operator()(u64 node) { return eql(node, air) ? 0 : 1; }
};

const std::vector<u64> ex = {0x0028000a00f08000,0x0028001200b8803b,0x0028001a006c7fff,0x0008001a00207fff,0x0007fff200108001,0x0027fff200088001,0x0027fff600048003,0x0017ffe5fffd8001,0x0017ffc9fffc8002,0x0027ffd600068001,0x0027ffe5fffe7fff,0x0008001a00217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0008001a00217fe5,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0008001200217ff8,0x0007fff200088001,0x0027fff200108002,0x0017ffe6000c8004,0x0027ffe6000e8001,0x0028001a000a7fff,0x0057ffc5fff47fff,0x0027ffc5fff57ffe,0x0017ff8600008001,0x0027fffa00018000,0x0017fd26000e8001,0x00280012002a7fff,0x00280012001c7fff,0x00080015fff47fff,0x0007fff200108001,0x0027fff6000c8001,0x0027fff600068001,0x00280015fffe7fff,0x0017ffc5fff47fff,0x0017ff9600008001,0x0027fffa00018000,0x0017ff6a00048001,0x0027fff600017fff,0x0027fc5200057fc4,0x0017fffa00048001,0x0027fff600017fff};  

int main(void) {
  // Creates net on host
  thrust::host_vector<u64> h_net(4096);
  thrust::fill(h_net.begin(), h_net.begin() + h_net.size(), air);
  for (int i = 0; i < ex.size(); ++i) h_net[i] = ex[i];
  h_net[128] = dot;
  thrust::host_vector<u64> h_pos(4096);
  thrust::sequence(h_pos.begin(), h_pos.end());

  /*for (u64 i = 0; i < 3; ++i) {*/
    /*reduce_pass(&h_net[0], h_net.size());*/
  /*}*/
  print_net(&h_net[0], h_net.size() / 8, false, true, true);

  // Sends to GPU
  thrust::device_vector<u64> d_net = h_net;
  thrust::device_vector<u64> d_pos = h_pos;

  // Reduces
  reduce_blocks<<<1,1024>>>(thrust::raw_pointer_cast(&d_net[0]), thrust::raw_pointer_cast(&d_pos[0]));

  // Sends to CPU & prints
  h_net = d_net;
  print_net(&h_net[0], h_net.size() / 8, false, true, true);

  // Tests if it tracked the dot position correctly.
  h_pos = d_pos;
  std::cout << "Dot position: " << h_pos[128] << " (" << (eql(h_net[h_pos[128]], dot) ? "correct" : "wrong") << ")" << std::endl;

  return 0;
}

